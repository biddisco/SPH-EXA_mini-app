/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! @file
 * @brief Benchmark cornerstone octree generation on the GPU
 *
 * @author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include <chrono>
#include <iostream>

#include <thrust/reduce.h>

#include "cstone/tree/octree.cuh"

#include "coord_samples/random.hpp"

using namespace cstone;

int main()
{
    using CodeType = unsigned;
    Box<double> box{-1, 1};

    int nParticles = 2000000;
    int bucketSize = 16;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    RandomGaussianCoordinates<double, CodeType> randomBox(nParticles, box);

    thrust::device_vector<CodeType> tree;
    thrust::device_vector<unsigned> counts;

    thrust::device_vector<CodeType> particleCodes(randomBox.mortonCodes().begin(),
                                                  randomBox.mortonCodes().end());

    hipEventRecord(start, hipStreamDefault);

    computeOctreeGpu(thrust::raw_pointer_cast(particleCodes.data()),
                     thrust::raw_pointer_cast(particleCodes.data() + nParticles),
                     bucketSize,
                     tree, counts);

    hipEventRecord(stop, hipStreamDefault);
    hipEventSynchronize(stop);

    float t0;
    hipEventElapsedTime(&t0, start, stop);
    std::cout << "build time from scratch " << t0/1000 << " nNodes(tree): " << nNodes(tree)
              << " count: " << thrust::reduce(counts.begin(), counts.end(), 0) << std::endl;

    hipEventRecord(start, hipStreamDefault);

    updateOctreeGpu(thrust::raw_pointer_cast(particleCodes.data()),
                    thrust::raw_pointer_cast(particleCodes.data() + nParticles),
                    bucketSize, tree, counts);

    hipEventRecord(stop, hipStreamDefault);
    hipEventSynchronize(stop);

    float t1;
    hipEventElapsedTime(&t1, start, stop);
    std::cout << "build time with guess " << t1/1000 << " nNodes(tree): " << nNodes(tree)
              << " count: " << thrust::reduce(counts.begin(), counts.end(), 0) << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}
