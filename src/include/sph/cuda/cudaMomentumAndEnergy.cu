#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "sph.cuh"
#include "utils.cuh"
#include "../kernels.hpp"

namespace sphexa
{
namespace sph
{
namespace cuda
{
template void computeMomentumAndEnergy<double, SqPatch<double>>(const std::vector<int> &clist, SqPatch<double> &d);

const double gradh_i = 1.0;
const double gradh_j = 1.0;
const double ep1 = 0.2, ep2 = 0.02;
const int mre = 4;

template <typename T>
__global__ void momenumAndEnergy(const int n, const int dx, const T sincIndex, const T K, const int ngmax, const BBox<T> *bbox,
                                 const int *clist, const int *neighbors, const int *neighborsCount, const T *x, const T *y, const T *z,
                                 const T *vx, const T *vy, const T *vz, const T *h, const T *m, const T *ro, const T *p, const T *c,
                                 T *grad_P_x, T *grad_P_y, T *grad_P_z, T *du)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= n) return;

    const int i = clist[tid];
    const int nn = neighborsCount[tid];
    T momentum_x = 0.0, momentum_y = 0.0, momentum_z = 0.0, energy = 0.0;

    T A_i = 0.0;
    if (p[i] < 0.0) A_i = 1.0;

    // int converstion to avoid a bug that prevents vectorization with some compilers
    for (int pj = 0; pj < nn; pj++)
    {
        const int j = neighbors[tid * ngmax + pj];

        // calculate the scalar product rv = rij * vij
        T r_ijx = (x[i] - x[j]);
        T r_ijy = (y[i] - y[j]);
        T r_ijz = (z[i] - z[j]);

        applyPBC(*bbox, 2.0 * h[i], r_ijx, r_ijy, r_ijz);

        const T v_ijx = (vx[i] - vx[j]);
        const T v_ijy = (vy[i] - vy[j]);
        const T v_ijz = (vz[i] - vz[j]);

        const T rv = r_ijx * v_ijx + r_ijy * v_ijy + r_ijz * v_ijz;

        const T r_square = (r_ijx * r_ijx) + (r_ijy * r_ijy) + (r_ijz * r_ijz);

        const T r_ij = sqrt(r_square);
        const T rv_i = r_ij / h[i];
        const T rv_j = r_ij / h[j];
        const T viscosity_ij = artificial_viscosity(ro[i], ro[j], h[i], h[j], c[i], c[j], rv, r_square);

        const T derivative_kernel_i = wharmonic_derivative_deprecated(rv_i, h[i], sincIndex, K);
        const T derivative_kernel_j = wharmonic_derivative_deprecated(rv_j, h[j], sincIndex, K);

        // divide by r_ij? missing h?
        const T grad_v_kernel_x_i = r_ijx * derivative_kernel_i;
        const T grad_v_kernel_y_i = r_ijy * derivative_kernel_i;
        const T grad_v_kernel_z_i = r_ijz * derivative_kernel_i;

        const T grad_v_kernel_x_j = r_ijx * derivative_kernel_j;
        const T grad_v_kernel_y_j = r_ijy * derivative_kernel_j;
        const T grad_v_kernel_z_j = r_ijz * derivative_kernel_j;

        const T grad_v_kernel_x_ij = (grad_v_kernel_x_i + grad_v_kernel_x_j) / 2.0;
        const T grad_v_kernel_y_ij = (grad_v_kernel_y_i + grad_v_kernel_y_j) / 2.0;
        const T grad_v_kernel_z_ij = (grad_v_kernel_z_i + grad_v_kernel_z_j) / 2.0;

        const T force_i_j_r = exp(-(rv_i * rv_i)) * exp((dx * dx) / (h[i] * h[i]));

        T A_j = 0.0;
        if (p[j] < 0.0) A_j = 1.0;

        T delta_pos_i_j = 0.0;
        if (p[i] > 0.0 && p[j] > 0.0) delta_pos_i_j = 1.0;

        const T R_i_j = ep1 * (A_i * abs(p[i]) + A_j * abs(p[j])) + ep2 * delta_pos_i_j * (abs(p[i]) + abs(p[j]));

        const T r_force_i_j = R_i_j * pow(force_i_j_r, (int)mre);

        const T partial_repulsive_force = (r_force_i_j / (ro[i] * ro[j]));

        const T pro_i = p[i] / (gradh_i * ro[i] * ro[i]);
        const T pro_j = p[j] / (gradh_j * ro[j] * ro[j]);

        momentum_x +=
            m[j] * (pro_i * grad_v_kernel_x_i + pro_j * grad_v_kernel_x_j + (partial_repulsive_force + viscosity_ij) * grad_v_kernel_x_ij);
        momentum_y +=
            m[j] * (pro_i * grad_v_kernel_y_i + pro_j * grad_v_kernel_y_j + (partial_repulsive_force + viscosity_ij) * grad_v_kernel_y_ij);
        momentum_z +=
            m[j] * (pro_i * grad_v_kernel_z_i + pro_j * grad_v_kernel_z_j + (partial_repulsive_force + viscosity_ij) * grad_v_kernel_z_ij);

        energy += m[j] * (pro_i + 0.5 * viscosity_ij) * (v_ijx * grad_v_kernel_x_i + v_ijy * grad_v_kernel_y_i + v_ijz * grad_v_kernel_z_i);
    }

    du[tid] = energy;

    grad_P_x[tid] = momentum_x;
    grad_P_y[tid] = momentum_y;
    grad_P_z[tid] = momentum_z;
}

template <typename T, class Dataset>
void computeMomentumAndEnergy(const std::vector<int> &clist, Dataset &d)
{
    const size_t n = clist.size();
    const size_t np = d.x.size();
    const size_t n_chunk = (size_t)n / d.noOfGpuLoopSplits;
    const size_t n_lastChunk =
        (size_t)n / d.noOfGpuLoopSplits + n % d.noOfGpuLoopSplits; // just in case n is not dividable by noOfGpuLoopSplits
    const size_t allNeighbors_chunk = n_chunk * d.ngmax;
    const size_t allNeighbors_lastChunk = n_lastChunk * d.ngmax;

    const size_t size_allNeighbors_chunk = allNeighbors_chunk * sizeof(int);
    const size_t size_allNeighbors_lastChunk = allNeighbors_lastChunk * sizeof(int);
    const size_t size_n_T_chunk = n_chunk * sizeof(T);
    const size_t size_n_T_lastChunk = n_lastChunk * sizeof(T);
    const size_t size_n_int_chunk = n_chunk * sizeof(int);
    const size_t size_n_int_lastChunk = n_lastChunk * sizeof(int);
    const size_t size_np_T = np * sizeof(T);
    const size_t size_bbox = sizeof(BBox<T>);

    int *d_clist, *d_neighbors, *d_neighborsCount; // d_ prefix stands for device
    T *d_x, *d_y, *d_z, *d_vx, *d_vy, *d_vz, *d_h, *d_m, *d_ro, *d_p, *d_c;
    T *d_grad_P_x, *d_grad_P_y, *d_grad_P_z, *d_du;
    BBox<T> *d_bbox;

    // const float neighborsSizeInGB = size_allNeighbors_chunk * 1e-9;
    // const float memorySizeInGB = (2 * size_n_int_chunk + size_allNeighbors_chunk + size_bbox + 11 * size_np_T_slice + 4 *
    // size_n_T_chunk)*1e-9; printf("CUDA: Total GPU memory usage: %.2fGB\n", memorySizeInGB);

    // input data
    utils::hipMalloc(size_n_int_lastChunk, d_clist, d_neighborsCount);
    utils::hipMalloc(size_allNeighbors_lastChunk, d_neighbors);
    utils::hipMalloc(size_bbox, d_bbox);
    utils::hipMalloc(size_np_T, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_m, d_ro, d_p, d_c);

    // output data
    utils::hipMalloc(size_n_T_lastChunk, d_grad_P_x, d_grad_P_y, d_grad_P_z, d_du);

    CHECK_CUDA_ERR(hipMemcpy(d_x, d.x.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_y, d.y.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_z, d.z.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_vx, d.vx.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_vy, d.vy.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_vz, d.vz.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_h, d.h.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_m, d.m.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_ro, d.ro.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_p, d.p.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_c, d.c.data(), size_np_T, hipMemcpyHostToDevice));
    CHECK_CUDA_ERR(hipMemcpy(d_bbox, &d.bbox, size_bbox, hipMemcpyHostToDevice));

    for (ushort s = 0; s < d.noOfGpuLoopSplits; ++s)
    {
        const int threadsPerBlock = 256;
        // printf("CUDA MomentumAndEnergy kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

        if (s == d.noOfGpuLoopSplits - 1) // if its the last chunk, send the rest of particle data to GPU
        {
            CHECK_CUDA_ERR(hipMemcpy(d_clist, clist.data() + (s * n_chunk), size_n_int_lastChunk, hipMemcpyHostToDevice));
            CHECK_CUDA_ERR(hipMemcpy(d_neighbors, d.neighbors.data() + (s * allNeighbors_chunk), size_allNeighbors_lastChunk,
                                      hipMemcpyHostToDevice));
            CHECK_CUDA_ERR(
                hipMemcpy(d_neighborsCount, d.neighborsCount.data() + (s * n_chunk), size_n_int_lastChunk, hipMemcpyHostToDevice));

            const int blocksPerGrid = (n_lastChunk + threadsPerBlock - 1) / threadsPerBlock;
            momenumAndEnergy<T><<<blocksPerGrid, threadsPerBlock>>>(n_lastChunk, d.dx, d.sincIndex, d.K, d.ngmax, d_bbox, d_clist,
                                                                    d_neighbors, d_neighborsCount, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h,
                                                                    d_m, d_ro, d_p, d_c, d_grad_P_x, d_grad_P_y, d_grad_P_z, d_du);

            CHECK_CUDA_ERR(hipGetLastError());

            CHECK_CUDA_ERR(hipMemcpy(d.grad_P_x.data() + (s * n_chunk), d_grad_P_x, size_n_T_lastChunk, hipMemcpyDeviceToHost));
            CHECK_CUDA_ERR(hipMemcpy(d.grad_P_y.data() + (s * n_chunk), d_grad_P_y, size_n_T_lastChunk, hipMemcpyDeviceToHost));
            CHECK_CUDA_ERR(hipMemcpy(d.grad_P_z.data() + (s * n_chunk), d_grad_P_z, size_n_T_lastChunk, hipMemcpyDeviceToHost));
            CHECK_CUDA_ERR(hipMemcpy(d.du.data() + (s * n_chunk), d_du, size_n_T_lastChunk, hipMemcpyDeviceToHost));
        }
        else
        {
            CHECK_CUDA_ERR(hipMemcpy(d_clist, clist.data() + (s * n_chunk), size_n_int_chunk, hipMemcpyHostToDevice));
            CHECK_CUDA_ERR(
                hipMemcpy(d_neighbors, d.neighbors.data() + (s * allNeighbors_chunk), size_allNeighbors_chunk, hipMemcpyHostToDevice));
            CHECK_CUDA_ERR(hipMemcpy(d_neighborsCount, d.neighborsCount.data() + (s * n_chunk), size_n_int_chunk, hipMemcpyHostToDevice));
            const int blocksPerGrid = (n_chunk + threadsPerBlock - 1) / threadsPerBlock;

            momenumAndEnergy<T><<<blocksPerGrid, threadsPerBlock>>>(n_chunk, d.dx, d.sincIndex, d.K, d.ngmax, d_bbox, d_clist, d_neighbors,
                                                                    d_neighborsCount, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_m, d_ro, d_p,
                                                                    d_c, d_grad_P_x, d_grad_P_y, d_grad_P_z, d_du);

            CHECK_CUDA_ERR(hipGetLastError());

            CHECK_CUDA_ERR(hipMemcpy(d.grad_P_x.data() + (s * n_chunk), d_grad_P_x, size_n_T_chunk, hipMemcpyDeviceToHost));
            CHECK_CUDA_ERR(hipMemcpy(d.grad_P_y.data() + (s * n_chunk), d_grad_P_y, size_n_T_chunk, hipMemcpyDeviceToHost));
            CHECK_CUDA_ERR(hipMemcpy(d.grad_P_z.data() + (s * n_chunk), d_grad_P_z, size_n_T_chunk, hipMemcpyDeviceToHost));
            CHECK_CUDA_ERR(hipMemcpy(d.du.data() + (s * n_chunk), d_du, size_n_T_chunk, hipMemcpyDeviceToHost));
        }
    }

    utils::hipFree(d_clist, d_neighborsCount, d_neighbors, d_bbox, d_x, d_y, d_z, d_vx, d_vy, d_vz, d_h, d_m, d_ro, d_p, d_c, d_grad_P_x,
                    d_grad_P_y, d_grad_P_z, d_du);
}
} // namespace cuda
} // namespace sph
} // namespace sphexa
