#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2021 CSCS, ETH Zurich
 *               2021 University of Basel
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */

/*! \file
 * \brief  Exposes gather functionality to reorder arrays by a map
 *
 * \author Sebastian Keller <sebastian.f.keller@gmail.com>
 */

#include <vector>

#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "gather.cuh"

template<class T, class I>
class DeviceMemory
{
public:

    DeviceMemory() = default;

    ~DeviceMemory()
    {
        if (allocatedSize_ > 0)
        {
            hipFree(d_ordering_);

            hipFree(d_buffer_[0]);
            hipFree(d_buffer_[1]);
        }
    }

    void reallocate(std::size_t newSize)
    {
        if (newSize > allocatedSize_)
        {
            // allocate 5% extra to avoid reallocation on small increase
            newSize = double(newSize) * 1.05;

            if (allocatedSize_ > 0)
            {
                hipFree(d_ordering_);

                hipFree(d_buffer_[0]);
                hipFree(d_buffer_[1]);
            }

            hipMalloc((void**)&d_ordering_,  newSize * sizeof(I));

            hipMalloc((void**)&(d_buffer_[0]), newSize * sizeof(T));
            hipMalloc((void**)&(d_buffer_[1]), newSize * sizeof(T));

            allocatedSize_ = newSize;
        }
    }

    I* ordering() { return d_ordering_; }

    T* deviceBuffer(int i)      { return d_buffer_[i]; }

private:
    std::size_t allocatedSize_{0} ;

    //! \brief reorder map
    I* d_ordering_;
    //! \brief device buffers
    T* d_buffer_[2];
};


template<class T, class I>
DeviceGather<T, I>::DeviceGather()
    : deviceMemory_(std::make_unique<DeviceMemory<T, I>>())
{}

template<class T, class I>
void DeviceGather<T, I>::setReorderMap(const I* map_first, const I* map_last)
{
    mapSize_      = map_last - map_first;
    deviceMemory_->reallocate(mapSize_);
    hipMemcpy(deviceMemory_->ordering(), map_first, mapSize_ * sizeof(I), hipMemcpyHostToDevice);
}

template<class T, class I>
DeviceGather<T, I>::~DeviceGather() = default;


template<class T, class I>
__global__ void reorder(I* map, T* source, T* destination, size_t n)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid > n) return;

    destination[tid] = source[map[tid]];
}

template<class T, class I>
void DeviceGather<T, I>::operator()(T* values)
{
    constexpr int nThreads = 256;
    int nBlocks = (mapSize_ + nThreads - 1) / nThreads;

    // upload to device
    hipMemcpy(deviceMemory_->deviceBuffer(0), values, mapSize_ * sizeof(T), hipMemcpyHostToDevice);

    // reorder on device
    reorder<<<nBlocks, nThreads>>>(deviceMemory_->ordering(),
                                   deviceMemory_->deviceBuffer(0),
                                   deviceMemory_->deviceBuffer(1),
                                   mapSize_);

    // download to host
    hipMemcpy(values, deviceMemory_->deviceBuffer(1), mapSize_ * sizeof(T), hipMemcpyDeviceToHost);
}

template class DeviceGather<float,  unsigned>;
template class DeviceGather<float,  uint64_t>;
template class DeviceGather<double, unsigned>;
template class DeviceGather<double, uint64_t>;
